#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
void CPURun()
{
    printf(" CPU work.\n");
}
__global__ void GPURun()
{
    printf("GPU work.\n");
}

int main()
{
    CPURun();
    GPURun << <1, 1 >> > ();
    hipDeviceSynchronize();
    return 0;
}